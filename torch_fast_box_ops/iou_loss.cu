#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

#include <cuda/cmath>

#include "iou_common.cuh"

auto loss_inter_union(const torch::Tensor &boxes1, const torch::Tensor &boxes2)
    -> std::tuple<torch::Tensor, torch::Tensor>
{
    TORCH_CHECK(boxes1.is_contiguous() && boxes2.is_contiguous(), "Input tensors must be contiguous");
    TORCH_CHECK(boxes1.sizes() == boxes2.sizes(), "Input tensors boxes1 and boxes2 must have the same shape");
    TORCH_CHECK(boxes1.ndimension() == 2 && boxes1.size(-1) == 4, "Input tensors must have shape (N, 4)");

    torch::Tensor intersection = boxes1.new_empty({ boxes1.size(0) });
    torch::Tensor union_area = boxes1.new_empty({ boxes1.size(0) });

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(boxes1.scalar_type(), "_loss_inter_union", [&] {
        const auto num_boxes = boxes1.size(0);
        const auto boxes1_ptr = static_cast<const XYXY<scalar_t> *>(boxes1.const_data_ptr());
        const auto boxes2_ptr = static_cast<const XYXY<scalar_t> *>(boxes2.const_data_ptr());
        auto intersection_ptr = static_cast<scalar_t *>(intersection.mutable_data_ptr());
        auto union_area_ptr = static_cast<scalar_t *>(union_area.mutable_data_ptr());

        if (boxes1.is_cuda()) {
            auto kernel = [=] __device__(unsigned int idx) {
                intersection_ptr[idx] = box_intersection_area(boxes1_ptr[idx], boxes2_ptr[idx]);
                union_area_ptr[idx] =
                    box_area_op(boxes1_ptr[idx]) + box_area_op(boxes2_ptr[idx]) - intersection_ptr[idx];
            };
            launch_elementwise_kernel(kernel, num_boxes, at::cuda::getCurrentCUDAStream());
        } else {
            for (std::size_t i = 0; i < num_boxes; ++i) {
                intersection_ptr[i] = box_intersection_area(boxes1_ptr[i], boxes2_ptr[i]);
                union_area_ptr[i] = box_area_op(boxes1_ptr[i]) + box_area_op(boxes2_ptr[i]) - intersection_ptr[i];
            }
        }
    });

    return { intersection, union_area };
}

template<typename T>
TFBO_HOST_DEVICE auto intersection_grad(const XYXY<T> &box1, const XYXY<T> &box2, const XYXY<T> &inter_box)
    -> std::tuple<XYXY<T>, XYXY<T>>
{
    XYXY<T> grad_box1, grad_box2;

    T inter_width = inter_box.x2 - inter_box.x1;
    T inter_height = inter_box.y2 - inter_box.y1;
    bool okay = inter_width > 0 && inter_height > 0;
    inter_width *= okay ? 1 : 0;
    inter_height *= okay ? 1 : 0;
    const auto subgrad = static_cast<T>(0.5);

    bool x1_gt = box1.x1 > box2.x1;
    bool x1_eq = box1.x1 == box2.x1;
    bool x1_lt = box1.x1 < box2.x1;
    grad_box1.x1 = -(x1_gt + subgrad * x1_eq) * inter_height;
    grad_box2.x1 = -(x1_lt + subgrad * x1_eq) * inter_height;

    bool y1_gt = box1.y1 > box2.y1;
    bool y1_eq = box1.y1 == box2.y1;
    bool y1_lt = box1.y1 < box2.y1;
    grad_box1.y1 = -(y1_gt + subgrad * y1_eq) * inter_width;
    grad_box2.y1 = -(y1_lt + subgrad * y1_eq) * inter_width;

    bool x2_gt = box1.x2 > box2.x2;
    bool x2_eq = box1.x2 == box2.x2;
    bool x2_lt = box1.x2 < box2.x2;
    grad_box1.x2 = (x2_lt + subgrad * x2_eq) * inter_height;
    grad_box2.x2 = (x2_gt + subgrad * x2_eq) * inter_height;

    bool y2_gt = box1.y2 > box2.y2;
    bool y2_eq = box1.y2 == box2.y2;
    bool y2_lt = box1.y2 < box2.y2;
    grad_box1.y2 = (y2_lt + subgrad * y2_eq) * inter_width;
    grad_box2.y2 = (y2_gt + subgrad * y2_eq) * inter_width;

    return { grad_box1, grad_box2 };
}

template<typename T>
TFBO_HOST_DEVICE auto inter_union_grad(T grad_inter, T grad_union, const XYXY<T> &box1, const XYXY<T> &box2)
    -> std::tuple<XYXY<T>, XYXY<T>>
{
    XYXY<T> inter_box = box_intersection(box1, box2);
    T inter_area = std::max(box_area_op(inter_box), static_cast<T>(0));
    T union_area = box_area_op(box1) + box_area_op(box2) - inter_area;

    auto [inter_grad_box1, inter_grad_box2] = intersection_grad(box1, box2, inter_box);
    auto area_grad_box1 = box_area_grad(box1);
    auto area_grad_box2 = box_area_grad(box2);

    // dUnion = dArea1 + dArea2 - dIntersection
    // grad = dUnion * gradUnion + dIntersection * gradInter
    // grad = (dArea - dIntersection) * gradUnion + dIntersection * gradInter
    // grad = dArea * gradUnion + (gradInter - gradUnion) * dIntersection
    T grad_inter_ = grad_inter - grad_union;

    XYXY<T> grad_box1;
    grad_box1.x1 = grad_inter_ * inter_grad_box1.x1 + grad_union * area_grad_box1.x1;
    grad_box1.y1 = grad_inter_ * inter_grad_box1.y1 + grad_union * area_grad_box1.y1;
    grad_box1.x2 = grad_inter_ * inter_grad_box1.x2 + grad_union * area_grad_box1.x2;
    grad_box1.y2 = grad_inter_ * inter_grad_box1.y2 + grad_union * area_grad_box1.y2;

    XYXY<T> grad_box2;
    grad_box2.x1 = grad_inter_ * inter_grad_box2.x1 + grad_union * area_grad_box2.x1;
    grad_box2.y1 = grad_inter_ * inter_grad_box2.y1 + grad_union * area_grad_box2.y1;
    grad_box2.x2 = grad_inter_ * inter_grad_box2.x2 + grad_union * area_grad_box2.x2;
    grad_box2.y2 = grad_inter_ * inter_grad_box2.y2 + grad_union * area_grad_box2.y2;

    return { grad_box1, grad_box2 };
}

auto loss_inter_union_backward(const torch::Tensor &grad_inter,
    const torch::Tensor &grad_union,
    const torch::Tensor &boxes1,
    const torch::Tensor &boxes2) -> std::tuple<torch::Tensor, torch::Tensor>
{
    TORCH_CHECK(
        grad_inter.is_contiguous() && grad_union.is_contiguous() && boxes1.is_contiguous() && boxes2.is_contiguous(),
        "Input tensors must be contiguous");
    TORCH_CHECK(boxes1.sizes() == boxes2.sizes(), "Input tensors boxes1 and boxes2 must have the same shape");
    TORCH_CHECK(boxes1.ndimension() == 2 && boxes1.size(-1) == 4, "Input tensors must have shape (N, 4)");

    auto grad_boxes1 = torch::empty_like(boxes1);
    auto grad_boxes2 = torch::empty_like(boxes2);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(boxes1.scalar_type(), "_loss_inter_union_backward", [&] {
        const auto num_boxes = boxes1.size(0);
        const auto boxes1_ptr = static_cast<const XYXY<scalar_t> *>(boxes1.const_data_ptr());
        const auto boxes2_ptr = static_cast<const XYXY<scalar_t> *>(boxes2.const_data_ptr());
        auto grad_boxes1_ptr = static_cast<XYXY<scalar_t> *>(grad_boxes1.mutable_data_ptr());
        auto grad_boxes2_ptr = static_cast<XYXY<scalar_t> *>(grad_boxes2.mutable_data_ptr());
        const auto grad_inter_ptr = grad_inter.const_data_ptr<scalar_t>();
        const auto grad_union_ptr = grad_union.const_data_ptr<scalar_t>();

        if (boxes1.is_cuda()) {
            auto kernel = [=] __device__(unsigned int idx) {
                auto [grad_boxes1, grad_boxes2] =
                    inter_union_grad(grad_inter_ptr[idx], grad_union_ptr[idx], boxes1_ptr[idx], boxes2_ptr[idx]);
                grad_boxes1_ptr[idx] = grad_boxes1;
                grad_boxes2_ptr[idx] = grad_boxes2;
            };
            launch_elementwise_kernel(kernel, num_boxes, at::cuda::getCurrentCUDAStream());
        } else {
            for (std::size_t i = 0; i < num_boxes; ++i) {
                std::tie(grad_boxes1_ptr[i], grad_boxes2_ptr[i]) =
                    inter_union_grad(grad_inter_ptr[i], grad_union_ptr[i], boxes1_ptr[i], boxes2_ptr[i]);
            }
        }
    });

    return { grad_boxes1, grad_boxes2 };
}

template<typename T> TFBO_HOST_DEVICE auto iou_loss_fn(const XYXY<T> &box1, const XYXY<T> &box2, T eps, giou_tag) -> T
{
    auto intersection = box_intersection_area(box1, box2);
    auto union_area = box_area_op(box1) + box_area_op(box2) - intersection;
    XYXY<T> enclosing_box = min_enclosing_box(box1, box2);
    T enclosing_area = std::max(box_area_op(enclosing_box), static_cast<T>(0));
    T giou = intersection / union_area - (enclosing_area - union_area) / (enclosing_area + eps);
    return 1 - giou;
}

template<typename T>
TFBO_HOST_DEVICE auto iou_grad(T grad_loss, const XYXY<T> &box1, const XYXY<T> &box2, T eps, giou_tag)
    -> std::tuple<XYXY<T>, XYXY<T>>
{
    T inter_area = box_intersection_area(box1, box2);
    T union_area = box_area_op(box1) + box_area_op(box2) - inter_area;
    XYXY<T> enclosing_box = min_enclosing_box(box1, box2);
    T enc_area = std::max(box_area_op(enclosing_box), static_cast<T>(0));

    T enc_area_eps = enc_area + eps;
    T union_area_eps = union_area + eps;

    T grad_enc_area = grad_loss * union_area / (enc_area_eps * enc_area_eps);
    T grad_inter = -grad_loss / union_area_eps;
    T grad_union = grad_loss * (inter_area / (union_area_eps * union_area_eps) - 1 / enc_area_eps);

    auto [grad_box1_enc, grad_box2_enc] = min_enclosing_box_grad(box1, box2, enclosing_box);
    auto [grad_box1, grad_box2] = inter_union_grad(grad_inter, grad_union, box1, box2);

    // Combine gradients with FMA
    grad_box1.x1 = fma(grad_box1_enc.x1, grad_enc_area, grad_box1.x1);
    grad_box1.y1 = fma(grad_box1_enc.y1, grad_enc_area, grad_box1.y1);
    grad_box1.x2 = fma(grad_box1_enc.x2, grad_enc_area, grad_box1.x2);
    grad_box1.y2 = fma(grad_box1_enc.y2, grad_enc_area, grad_box1.y2);

    grad_box2.x1 = fma(grad_box2_enc.x1, grad_enc_area, grad_box2.x1);
    grad_box2.y1 = fma(grad_box2_enc.y1, grad_enc_area, grad_box2.y1);
    grad_box2.x2 = fma(grad_box2_enc.x2, grad_enc_area, grad_box2.x2);
    grad_box2.y2 = fma(grad_box2_enc.y2, grad_enc_area, grad_box2.y2);

    return { grad_box1, grad_box2 };
}

template<typename T> TFBO_HOST_DEVICE auto iou_loss_fn(const XYXY<T> &box1, const XYXY<T> &box2, T eps, diou_tag) -> T
{
    auto intersection = box_intersection_area(box1, box2);
    auto union_area = box_area_op(box1) + box_area_op(box2) - intersection;
    XYXY<T> enclosing_box = min_enclosing_box(box1, box2);
    const T diag_dist_sq = dist_sq<T>(enclosing_box.x2 - enclosing_box.x1, enclosing_box.y2 - enclosing_box.y1);
    const CXCY<T> box1c(box1);
    const CXCY<T> box2c(box2);
    const T cent_dist_sq = dist_sq<T>(box1c.cx - box2c.cx, box1c.cy - box2c.cy);
    return 1 - intersection / union_area + cent_dist_sq / (diag_dist_sq + static_cast<T>(1e-7));
}

/**
 * @brief  Gradient of a box corner p with respect to center distance squared value C -> dC/dp
 *         is a function of the box corner (x1), its opposite corner (x2) and the other box's center point (cx)
 *
 * @example gradient of box1 x1 is cdist_grad(box1.x1, box1.x2, box2.cx)
 *
 * @tparam T type of box points
 */
template<typename T> TFBO_HOST_DEVICE auto cdist_grad(T p1, T p2, T p3) -> T { return 0.5 * (p1 + p2) - p3; }

template<typename T>
TFBO_HOST_DEVICE auto iou_grad(T grad_loss, const XYXY<T> &box1, const XYXY<T> &box2, T eps, diou_tag)
    -> std::tuple<XYXY<T>, XYXY<T>>
{
    const T inter_area = box_intersection_area(box1, box2);
    const T union_area = box_area_op(box1) + box_area_op(box2) - inter_area;
    const XYXY enclosing_box = min_enclosing_box(box1, box2);
    const CXCY box1c(box1);
    const CXCY box2c(box2);
    const T diag_dist_sq = dist_sq<T>(enclosing_box.x2 - enclosing_box.x1, enclosing_box.y2 - enclosing_box.y1);
    const T cent_dist_sq = dist_sq<T>(box1c.cx - box2c.cx, box1c.cy - box2c.cy);

    const T union_area_eps = union_area + eps;
    const T grad_inter = -grad_loss / union_area_eps;
    const T grad_union = grad_loss * inter_area / (union_area_eps * union_area_eps);

    T grad_cent_dist = grad_loss / diag_dist_sq;
    T grad_diag_dist = -grad_loss * cent_dist_sq / (diag_dist_sq * diag_dist_sq);

    auto [grad_box1, grad_box2] = inter_union_grad(grad_inter, grad_union, box1, box2);
    return { grad_box1, grad_box2 };
}

template<typename IoUType>
auto box_iou_loss(const torch::Tensor &boxes1, const torch::Tensor &boxes2, double eps) -> torch::Tensor
{
    TORCH_CHECK(boxes1.is_contiguous() && boxes2.is_contiguous(), "Input tensors must be contiguous");
    TORCH_CHECK(boxes1.sizes() == boxes2.sizes(), "Input tensors boxes1 and boxes2 must have the same shape");
    TORCH_CHECK(boxes1.ndimension() == 2 && boxes1.size(-1) == 4, "Input tensors must have shape (N, 4)");

    auto loss = boxes1.new_empty({ boxes1.size(0) });

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(boxes1.scalar_type(), "box_iou_loss", [&] {
        const auto num_boxes = boxes1.size(0);
        const auto boxes1_ptr = static_cast<const XYXY<scalar_t> *>(boxes1.const_data_ptr());
        const auto boxes2_ptr = static_cast<const XYXY<scalar_t> *>(boxes2.const_data_ptr());
        auto loss_ptr = loss.mutable_data_ptr<scalar_t>();
        const auto eps_t = static_cast<scalar_t>(eps);

        if (boxes1.is_cuda()) {
            auto kernel = [=] __device__(unsigned int idx) {
                loss_ptr[idx] = iou_loss_fn(boxes1_ptr[idx], boxes2_ptr[idx], eps_t, IoUType{});
            };
            launch_elementwise_kernel(kernel, num_boxes, at::cuda::getCurrentCUDAStream());
        } else {
            for (std::size_t i = 0; i < num_boxes; ++i) {
                loss_ptr[i] = iou_loss_fn(boxes1_ptr[i], boxes2_ptr[i], eps_t, IoUType{});
            }
        }
    });

    return loss;
}

template<typename IoUType>
auto box_iou_loss_backward(const torch::Tensor &grad,
    const torch::Tensor &boxes1,
    const torch::Tensor &boxes2,
    double eps) -> std::tuple<torch::Tensor, torch::Tensor>
{
    TORCH_CHECK(
        grad.is_contiguous() && boxes1.is_contiguous() && boxes2.is_contiguous(), "Input tensors must be contiguous");
    TORCH_CHECK(boxes1.sizes() == boxes2.sizes(), "Input tensors boxes1 and boxes2 must have the same shape");
    TORCH_CHECK(boxes1.ndimension() == 2 && boxes1.size(-1) == 4, "Input tensors must have shape (N, 4)");

    auto grad_boxes1 = torch::empty_like(boxes1);
    auto grad_boxes2 = torch::empty_like(boxes2);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(boxes1.scalar_type(), "box_iou_loss_backward", [&] {
        const auto num_boxes = boxes1.size(0);
        const auto boxes1_ptr = static_cast<const XYXY<scalar_t> *>(boxes1.const_data_ptr());
        const auto boxes2_ptr = static_cast<const XYXY<scalar_t> *>(boxes2.const_data_ptr());
        auto grad_boxes1_ptr = static_cast<XYXY<scalar_t> *>(grad_boxes1.mutable_data_ptr());
        auto grad_boxes2_ptr = static_cast<XYXY<scalar_t> *>(grad_boxes2.mutable_data_ptr());
        const auto grad_ptr = grad.const_data_ptr<scalar_t>();
        auto eps_t = static_cast<scalar_t>(eps);
        if (boxes1.is_cuda()) {
            auto kernel = [=] __device__(unsigned int idx) {
                auto [grad_box1, grad_box2] =
                    iou_grad(grad_ptr[idx], boxes1_ptr[idx], boxes2_ptr[idx], eps_t, IoUType{});
                grad_boxes1_ptr[idx] = grad_box1;
                grad_boxes2_ptr[idx] = grad_box2;
            };
            launch_elementwise_kernel(kernel, num_boxes, at::cuda::getCurrentCUDAStream());
        } else {
            for (std::size_t i = 0; i < num_boxes; ++i) {
                std::tie(grad_boxes1_ptr[i], grad_boxes2_ptr[i]) =
                    iou_grad(grad_ptr[i], boxes1_ptr[i], boxes2_ptr[i], eps_t, IoUType{});
            }
        }
    });

    return { grad_boxes1, grad_boxes2 };
}

template<typename T>
TFBO_HOST_DEVICE auto min_enclosing_box_grad(const XYXY<T> &box1, const XYXY<T> &box2, const XYXY<T> &enc_box)
    -> std::tuple<XYXY<T>, XYXY<T>>
{
    XYXY<T> box1_grad, box2_grad;
    T enc_w = enc_box.x2 - enc_box.x1;
    T enc_h = enc_box.y2 - enc_box.y1;
    const T subgrad = static_cast<T>(0.5);

    bool x1_lt = box1.x1 < box2.x1;
    bool x1_eq = box1.x1 == box2.x1;
    bool x1_gt = box1.x1 > box2.x1;
    box1_grad.x1 = -(x1_lt + subgrad * x1_eq) * enc_h;
    box2_grad.x1 = -(x1_gt + subgrad * x1_eq) * enc_h;

    bool y1_lt = box1.y1 < box2.y1;
    bool y1_eq = box1.y1 == box2.y1;
    bool y1_gt = box1.y1 > box2.y1;
    box1_grad.y1 = -(y1_lt + subgrad * y1_eq) * enc_w;
    box2_grad.y1 = -(y1_gt + subgrad * y1_eq) * enc_w;

    bool x2_lt = box1.x2 < box2.x2;
    bool x2_eq = box1.x2 == box2.x2;
    bool x2_gt = box1.x2 > box2.x2;
    box1_grad.x2 = (x2_gt + subgrad * x2_eq) * enc_h;
    box2_grad.x2 = (x2_lt + subgrad * x2_eq) * enc_h;

    bool y2_lt = box1.y2 < box2.y2;
    bool y2_eq = box1.y2 == box2.y2;
    bool y2_gt = box1.y2 > box2.y2;
    box1_grad.y2 = (y2_gt + subgrad * y2_eq) * enc_w;
    box2_grad.y2 = (y2_lt + subgrad * y2_eq) * enc_w;

    return { box1_grad, box2_grad };
}


TORCH_LIBRARY_IMPL(box_ops, CPU, m)
{
    m.impl("_loss_inter_union", &loss_inter_union);
    m.impl("_loss_inter_union_backward", &loss_inter_union_backward);
    m.impl("generalized_box_iou_loss", &box_iou_loss<giou_tag>);
    m.impl("generalized_box_iou_loss_backward", &box_iou_loss_backward<giou_tag>);
    m.impl("distance_box_iou_loss", &box_iou_loss<diou_tag>);
    m.impl("distance_box_iou_loss_backward", &box_iou_loss_backward<diou_tag>);
}

TORCH_LIBRARY_IMPL(box_ops, CUDA, m)
{
    m.impl("_loss_inter_union", &loss_inter_union);
    m.impl("_loss_inter_union_backward", &loss_inter_union_backward);
    m.impl("generalized_box_iou_loss", &box_iou_loss<giou_tag>);
    m.impl("generalized_box_iou_loss_backward", &box_iou_loss_backward<giou_tag>);
    m.impl("distance_box_iou_loss", &box_iou_loss<diou_tag>);
    m.impl("distance_box_iou_loss_backward", &box_iou_loss_backward<diou_tag>);
}
